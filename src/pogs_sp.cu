#include "hip/hip_runtime.h"
#include <cmath>
#include <algorithm>
#include <vector>

#include "_interface_defs.h"
#include "cml/cgls.h"
#include "cml/cml_blas.h"
#include "cml/cml_linalg.h"
#include "cml/cml_csrmat.h"
#include "cml/cml_vector.h"
#include "pogs.h"
#include "sinkhorn_knopp.h"

// Proximal Operator Graph Solver.
template<typename T, typename M>
int Pogs(PogsData<T, M> *pogs_data) {
  // Constants for adaptive-rho and over-relaxation.
  const T kDeltaMin = static_cast<T>(1.05);
  const T kGamma = static_cast<T>(1.01);
  const T kTau = static_cast<T>(0.8);
  const T kAlpha = static_cast<T>(1.7);
  const T kKappa = static_cast<T>(0.9);
  const T kOne = static_cast<T>(1);
  const T kZero = static_cast<T>(0);

  int err = 0;

  // Extract values from pogs_data
  size_t m = pogs_data->m, n = pogs_data->n, min_dim = std::min(m, n);
  T rho = pogs_data->rho;
  thrust::device_vector<FunctionObj<T> > f = pogs_data->f;
  thrust::device_vector<FunctionObj<T> > g = pogs_data->g;

  // Create cuBLAS hdl.
  hipblasHandle_t b_hdl;
  hipblasCreate(&hdl);
  hipsparseHandle_t s_hdl;
  hipsparseCreate(&shdl);

  // Allocate data for ADMM variables.
  bool pre_process = true;
  cml::vector<T> de, z, zt;
  cml::vector<T> zprev = cml::vector_calloc<T>(m + n);
  cml::vector<T> z12 = cml::vector_calloc<T>(m + n);
  cml::spmat<T, M::Fmt> A;
  if (pogs_data->factors.val != 0) {
    hipMemcpy(&rho, pogs_data->factors.val, sizeof(T), hipMemcpyDeviceToHost);
    pre_process = (rho == 0);
    if (pre_process)
      rho = pogs_data->rho;
    de = cml::vector_view_array(pogs_data->factors.val + 1, m + n);
    z = cml::vector_view_array(pogs_data->factors.val + 1 + m + n, m + n);
    zt = cml::vector_view_array(pogs_data->factors.val + 1 + 2 * (m + n),
        m + n);
    A = cml::spmat<T, M::Fmt>(pogs_data->factors.val + 1 + 3 * (m + n),
        pogs_data->factors.ind, pogs_data->factors.ptr, m, n,
        pogs_data->factors.nnz);
  } else {
    de = cml::vector_calloc<T>(m + n);
    z = cml::vector_calloc<T>(m + n);
    zt = cml::vector_calloc<T>(m + n);
    A = cml::spmat_alloc<T, M::Fmt>(m, n);
  }

  if (de.data == 0 || z.data == 0 || zt.data == 0 || zprev.data == 0 ||
      z12.data == 0 || A.val == 0 || A.ind == 0 || A.ptr == 0)
    err = 1;

  // Create views for x and y components.
  cml::vector<T> d = cml::vector_subvector(&de, 0, m);
  cml::vector<T> e = cml::vector_subvector(&de, m, n);
  cml::vector<T> x = cml::vector_subvector(&z, 0, n);
  cml::vector<T> y = cml::vector_subvector(&z, n, m);
  cml::vector<T> x12 = cml::vector_subvector(&z12, 0, n);
  cml::vector<T> y12 = cml::vector_subvector(&z12, n, m);

  if (pre_process && !err) {
    // Copy A to device (assume input row-major).
    cml::spmat_memcpy(&A, pogs_data->A.val, pogs_data->A.ind, pogs_data->A.ptr);
    err = Equilibrate(&A, &d, &e);

    // Scale f and g to account for diagonal scaling e and d.
    if (!err) {
      thrust::transform(f.begin(), f.end(), thrust::device_pointer_cast(d.data),
          f.begin(), ApplyOp<T, thrust::divides<T> >(thrust::divides<T>()));
      thrust::transform(g.begin(), g.end(), thrust::device_pointer_cast(e.data),
          g.begin(), ApplyOp<T, thrust::multiplies<T> >(thrust::multiplies<T>()));
    }
  }

  // Signal start of execution.
  if (!pogs_data->quiet)
    Printf("   #      res_pri    eps_pri   res_dual   eps_dual"
           "        gap    eps_gap  objective\n");

  // Initialize scalars.
  T sqrtn_atol = std::sqrt(static_cast<T>(n)) * pogs_data->abs_tol;
  T sqrtm_atol = std::sqrt(static_cast<T>(m)) * pogs_data->abs_tol;
  T delta = kDeltaMin, xi = static_cast<T>(1.0);
  unsigned int kd = 0, ku = 0;
  bool converged = false;

  for (unsigned int k = 0; k < pogs_data->max_iter && !err; ++k) {
    cml::vector_memcpy(&zprev, &z);

    // Evaluate Proximal Operators
    cml::blas_axpy(b_hdl, -kOne, &zt, &z);
    ProxEval(g, rho, x.data, x.stride, x12.data, x12.stride);
    ProxEval(f, rho, y.data, y.stride, y12.data, y12.stride);

    // Compute dual variable.
    T nrm_r = 0, nrm_s = 0;
    cml::blas_axpy(b_hdl, -kOne, &z12, &z);
    cml::blas_dot(b_hdl, &z, &z12, &gap);
    pogs_data->optval = FuncEval(f, y12.data, 1) + FuncEval(g, x12.data, 1);
    T eps_pri = sqrtm_atol + pogs_data->rel_tol * cml::blas_nrm2(b_hdl, &z12);
    T eps_dua = sqrtn_atol + pogs_data->rel_tol * rho * cml::blas_nrm2(b_hdl, &z);

    if (converged)
      break;

    // Project and Update Dual Variables


    // Apply over relaxation.
    cml::blas_scal(b_hdl, kAlpha, &z);
    cml::blas_axpy(b_hdl, kOne - kAlpha, &zprev, &z);

    // Update dual variable.
    cml::blas_axpy(b_hdl, kAlpha, &z12, &zt);
    cml::blas_axpy(b_hdl, kOne - kAlpha, &zprev, &zt);
    cml::blas_axpy(b_hdl, -kOne, &z, &zt);

    bool exact = false;
    if (m >= n) {
      cml::vector_memcpy(&zprev, &z12);
      cml::blas_axpy(b_hdl, -kOne, &z, &zprev);
      nrm_r = cml::blas_nrm2(b_hdl, &zprev);
      if (nrm_s < eps_dua && nrm_r < eps_pri) {
        cml::blas_gemv(b_hdl, HIPBLAS_OP_N, kOne, &A, &x12, -kOne, &y12);
        nrm_r = cml::blas_nrm2(b_hdl, &y12);
        exact = true;
      }
    } else {
      cml::blas_axpy(b_hdl, -kOne, &zprev, &z12);
      cml::blas_axpy(b_hdl, -kOne, &z, &zprev);
      nrm_s = rho * cml::blas_nrm2(b_hdl, &zprev);
      if (nrm_r < eps_pri && nrm_s < eps_dua) {
        cml::blas_gemv(b_hdl, HIPBLAS_OP_T, kOne, &A, &y12, kOne, &x12);
        nrm_s = rho * cml::blas_nrm2(b_hdl, &x12);
        exact = true;
      }
    }

    // Evaluate stopping criteria.
    converged = exact && nrm_r < eps_pri && nrm_s < eps_dua && gap < eps_gap;
    if (!pogs_data->quiet && (k % 10 == 0 || converged))
      Printf("%4d :  %.3e  %.3e  %.3e  %.3e  %.3e  %.3e  %.3e\n",
          k, nrm_r, eps_pri, nrm_s, eps_dua, gap, eps_gap, pogs_data->optval);

    // Rescale rho.
    if (pogs_data->adaptive_rho && !converged) {
      if (nrm_s < xi * eps_dua && nrm_r > xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(kd)) {
        rho *= delta;
        cml::blas_scal(b_hdl, 1 / delta, &zt);
        delta = std::min(kGamma * delta, kDeltaMax);
        ku = k;
      } else if (nrm_s > xi * eps_dua && nrm_r < xi * eps_pri &&
          kTau * static_cast<T>(k) > static_cast<T>(ku)) {
        rho /= delta;
        cml::blas_scal(b_hdl, delta, &zt);
        delta = std::min(kGamma * delta, kDeltaMax);
        kd = k;
      } else if (nrm_s < xi * eps_dua && nrm_r < xi * eps_pri) {
        xi *= kKappa;
      } else {
        delta = std::max(delta / kGamma, kDeltaMin);
      }
    }
  }

}
