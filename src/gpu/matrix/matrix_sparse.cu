#include <assert.h>
#include <hipblas.h>
#include <hipsparse.h>

#include <memory>

#include "cml/cml_spblas.cuh"
#include "cml/cml_spmat.cuh"
#include "cml/cml_vector.cuh"
#include "matrix/matrix.h"
#include "matrix/matrix_sparse.h"
#include "util.cuh"

namespace pogs {

namespace {

template <typename T>
struct GpuData {
  const T *orig_data;
  const POGS_INT *orig_ptr;
  const POGS_INT *orig_ind;
  hipblasHandle_t d_hdl;
  hipsparseHandle_t s_hdl;
  hipsparseMatDescr_t descr;
  GpuData(const T *data, const POGS_INT *ptr, const POGS_INT *ind)
      : orig_data(data), orig_ptr(ptr), orig_ind(ind) {
    hipblasCreate(&d_hdl);
    hipsparseCreate(&s_hdl);
    hipsparseCreateMatDescr(&descr);
    DEBUG_CUDA_CHECK_ERR();
  }
  ~GpuData() {
    hipblasDestroy(d_hdl);
    hipsparseDestroy(s_hdl);
    hipsparseDestroyMatDescr(descr);
    DEBUG_CUDA_CHECK_ERR();
  }
};

hipsparseOperation_t OpToCusparseOp(char trans) {
  assert(trans == 'n' || trans == 'N' || trans == 't' || trans == 'T');
  return trans == 'n' || trans == 'N'
      ? HIPSPARSE_OPERATION_NON_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE;
}

}  // namespace

template <typename T>
MatrixSparse<T>::MatrixSparse(char ord, POGS_INT m, POGS_INT n, POGS_INT nnz,
                              const T *data, const POGS_INT *ptr,
                              const POGS_INT *ind)
    : Matrix<T>(m, n), _data(0), _ptr(0), _ind(0), _nnz(nnz) {
  assert(ord == 'r' || ord == 'R' || ord == 'c' || ord == 'C');
  _ord = (ord == 'r' || ord == 'R') ? ROW : COL;

  // Set GPU specific data.
  GpuData<T> *info = new GpuData<T>(data, ptr, ind);
  this->_info = reinterpret_cast<void*>(info);
}

template <typename T>
MatrixSparse<T>::MatrixSparse(const MatrixSparse<T>& A)
    : Matrix<T>(A._m, A._n), _data(0), _ptr(0), _ind(0), _nnz(A._nnz), 
      _ord(A._ord) {

  GpuData<T> *info_A = reinterpret_cast<GpuData<T>*>(A._info);
  GpuData<T> *info = new GpuData<T>(info_A->orig_data, info_A->orig_ptr,
      info_A->orig_ind);
  this->_info = reinterpret_cast<void*>(info);
}

template <typename T>
MatrixSparse<T>::~MatrixSparse() {
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  delete info;
  this->_info = 0;

  if (this->_done_init) {
    if (_data) {
      hipFree(_data);
      _data = 0;
      DEBUG_CUDA_CHECK_ERR();
    }

    if (_ptr) {
      hipFree(_ptr);
      _ptr = 0;
      DEBUG_CUDA_CHECK_ERR();
    }

    if (_ind) {
      hipFree(_ind);
      _ind = 0;
      DEBUG_CUDA_CHECK_ERR();
    }
  }
}

template <typename T>
int MatrixSparse<T>::Init() {
  DEBUG_ASSERT(!this->_done_init);
  if (this->_done_init)
    return 1;
  this->_done_init = true;

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);

  // Allocate sparse matrix on gpu.
  hipMalloc(&_data, 2 * _nnz * sizeof(T));
  hipMalloc(&_ind, 2 * _nnz * sizeof(POGS_INT));
  hipMalloc(&_ptr, (this->_m + this->_n + 2) * sizeof(POGS_INT));
  DEBUG_CUDA_CHECK_ERR();

  if (_ord == ROW) {
    cml::spmat<T, POGS_INT, CblasRowMajor> A(_data, _ptr, _ind, this->_m,
        this->_n, _nnz);
    cml::spmat_memcpy(info->s_hdl, &A, _data, _ind, _ptr);
  } else {
    cml::spmat<T, POGS_INT, CblasColMajor> A(_data, _ptr, _ind, this->_m,
        this->_n, _nnz);
    cml::spmat_memcpy(info->s_hdl, &A, _data, _ind, _ptr);
  }
  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixSparse<T>::Mul(char trans, T alpha, const T *x, T beta, T *y) const {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);

  const cml::vector<T> x_vec = cml::vector_view_array<T>(x, this->_n);
  cml::vector<T> y_vec = cml::vector_view_array<T>(y, this->_m);

  if (_ord == ROW) {
    cml::spmat<T, POGS_INT, CblasRowMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  } else {
    cml::spmat<T, POGS_INT, CblasColMajor> A(_data, _ind, _ptr, this->_m,
        this->_n, _nnz);
    cml::spblas_gemv(info->s_hdl, OpToCusparseOp(trans), info->descr, alpha,
        &A, &x_vec, beta, &y_vec);
  }
  DEBUG_CUDA_CHECK_ERR();

  return 0;
}

template <typename T>
int MatrixSparse<T>::Equil(T *d, T *e) {
  DEBUG_ASSERT(this->_done_init);
  if (!this->_done_init)
    return 1;

  cml::vector<T> d_vec = cml::vector_view_array<T>(d, this->_m);
  cml::vector<T> e_vec = cml::vector_view_array<T>(e, this->_n);

  // TODO: implement proper equilibration
  cml::vector_set_all(&d_vec, static_cast<T>(1));
  cml::vector_set_all(&e_vec, static_cast<T>(1));

  return 0;
}

template class MatrixSparse<double>;
template class MatrixSparse<float>;

}  // namespace pogs

