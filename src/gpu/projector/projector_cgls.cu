#include "hip/hip_runtime.h"
#include <hipblas.h>

#include <algorithm>
#include <limits>

#include "cgls.cuh"
#include "cml/cml_blas.cuh"
#include "projector/projector_cgls.h"
#include "projector_helper.cuh"
#include "util.cuh"

namespace pogs {

namespace {

template<typename T>
struct GpuData {
  hipblasHandle_t handle;
  GpuData() {
    hipblasCreate(&handle);
    CUDA_CHECK_ERR();
  }
  ~GpuData() {
    hipblasDestroy(handle);
    CUDA_CHECK_ERR();
  }
};

}  // namespace

template <typename T, typename M>
ProjectorCgls<T, M>::ProjectorCgls(const M& A)
    : _A(A) {
  // Set GPU specific this->_info.
  GpuData<T> *info = new GpuData<T>();
  this->_info = reinterpret_cast<void*>(info);
}

ProjectorCgls<T, M>::~ProjectorCgls() {
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  delete info;
  this->_info = 0;
}

template <typename T, typename M>
int ProjectorCgls<T, M>::Init() {
  if (this->_done_init)
    return 1;
  this->_done_init = true;

  ASSERT(_A.IsInit());

  return 0;
}

template <typename T, typename M>
int ProjectorCgls<T, M>::Project(const T *x0, const T *y0, T s, T *x, T *y) {
  DEBUG_EXPECT(this->_done_init);
  DEBUG_EXPECT(s >= static_cast<T>(0.));
  if (!this->_done_init || s < static_cast<T>(0.))
    return 1;

  // Get Cublas handle
  GpuData<T> *info = reinterpret_cast<GpuData<T>*>(this->_info);
  hipblasHandle_t hdl = info->handle;

  // CGLS Gemv struct for matrix multiplication.
  struct Gemv : cgls::Gemv<T> {
    int operator()(char op, const T alpha, const T *x, const T beta, T *y) {
      return _A.Mul(op, alpha, x, beta, y);
    }
  };

  // Set initial y and x.
  hipMemcpy(y, y0, _A.Rows() * sizeof(T));
  hipMemset(x, 0, _A.Cols() * sizeof(T));

  // y := y0 - Ax0;
  _A.Mul('n', static_cast<T>(-1.), x0, static_cast<T>(1.), y);

  // Minimize ||Ax - b||_2^2 + s||x||_2^2
  cgls::Solve(hdl, Gemv(), _A.Rows(), _A.Cols(), y, x, s, kTol, kMaxIter, true);
  hipDeviceSynchronize();
 
  // x := x - x0
  cml::vector<T> x_vec = cml::vector_view_array(x, _A.Cols());
  cml::vector<T> x0_vec = cml::vector_view_array(x0, _A.Cols());
  cml::blas_axpy(hdl, static_cast<T>(-1.), x0_vec, x_vec);
  hipDeviceSynchronize();

  // y := Ax
  _A.Mul('n', static_cast<T>(1.), x, static_cast<T>(0.), y);
  hipDeviceSynchronize();

#ifdef DEBUG
  // Verify that projection was successful.
  CheckProjection(&_A, x, y, s);
#endif

  return 0;
}

template class ProjectorCgls<double, MatrixDense<double> >;
template class ProjectorCgls<float, MatrixDense<float> >;

template class ProjectorCgls<double, MatrixSparse<double> >;
template class ProjectorCgls<float, MatrixSparse<float> >;

}  // namespace pogs

