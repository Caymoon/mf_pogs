#include "hip/hip_runtime.h"
//  CGLS Conjugate Gradient Least Squares
//  Attempts to solve the system of linear equations A * x = b
//  for x.
//
//  Template Arguments:
//  T          - Data type (float or double).
//
//  F          - Sparse ordering (CSC or CSR).
//
//  Function Arguments:
//  handle_s   - Cusparse handle.
// 
//  handle_b   - Cublas handle.
//
//  descr      - Cusparse matrix descriptor (i.e. 0- or 1- based indexing)
//
//  val        - Array of matrix values.
//
//  ptr        - Column pointer if (F is CSC) or row poitner if (F is CSR).
//
//  ind        - Row indices if (F is CSC) or column indices if (F is CSR).
//
//  (m, n)     - Matrix dimensions of A.
//
//  nnz        - Number of non-zeros in A.
//
//  b          - Pointer to right-hand-side vector.
//
//  x          - Pointer to solution and initial guess.
//
//  shift      - Regularization parameter. Solves (A'*A + shift*I)*x = A'*b.
//
//  tol        - Specifies tolerance (recommended 1e-6).
//
//  maxit      - Maximum number of iterations (recommended 20).
//
//  quiet      - Disable printing to console.
//
//  Returns:
//  1. CGLS converged to the desired tolerance tol within maxit iterations.
//  2. CGLS iterated maxit times but did not converge.
//  3. Matrix (A'*A + shift*I) seems to be singular or indefinite.
//  4. Likely instable, (A'*A + shift*I) indefinite and norm(x) decreased.
//
//  Reference:
//  http://web.stanford.edu/group/SOL/software/cgls/
//

#include <stdio.h>

#include <algorithm>

#include <hipblas.h>
#include <hipsparse.h>

namespace cgls {

typedef int INT;

enum CGLS_FMT { CSC, CSR };

// Sparse matrix-vector multiply
template <typename T, CGLS_FMT F>
hipsparseStatus_t spmv(hipsparseHandle_t handle, hipsparseOperation_t transA,
                      INT m, INT n, INT nnz, const T *alpha,
                      hipsparseMatDescr_t descrA, const T *val, const INT *ptr,
                      const INT *ind, const T *x, const T *beta, T *y);

template <>
hipsparseStatus_t spmv<double, CSR>(hipsparseHandle_t handle,
                                   hipsparseOperation_t transA, INT m, INT n,
                                   INT nnz, const double *alpha,
                                   hipsparseMatDescr_t descrA, const double *val,
                                   const INT *ptr, const INT *ind,
                                   const double *x, const double *beta,
                                   double *y) {
  return hipsparseDcsrmv(handle, transA, static_cast<int>(m),
      static_cast<int>(n), static_cast<int>(nnz), alpha, descrA, val,
      static_cast<const int*>(ptr), static_cast<const int*>(ind), x, beta, y);
}

template <>
hipsparseStatus_t spmv<double, CSC>(hipsparseHandle_t handle,
                                   hipsparseOperation_t transA, INT m, INT n,
                                   INT nnz, const double *alpha,
                                   hipsparseMatDescr_t descrA, const double *val,
                                   const INT *ptr, const INT *ind,
                                   const double *x, const double *beta,
                                   double *y) {
  if (transA ==	HIPSPARSE_OPERATION_TRANSPOSE)
    transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else
    transA = HIPSPARSE_OPERATION_TRANSPOSE;
  return hipsparseDcsrmv(handle, transA, static_cast<int>(n),
      static_cast<int>(m), static_cast<int>(nnz), alpha, descrA, val,
      static_cast<const int*>(ptr), static_cast<const int*>(ind), x, beta, y);
}

template <>
hipsparseStatus_t spmv<float, CSR>(hipsparseHandle_t handle,
                                  hipsparseOperation_t transA, INT m, INT n,
                                  INT nnz, const float *alpha,
                                  hipsparseMatDescr_t descrA, const float *val,
                                  const INT *ptr, const INT *ind,
                                  const float *x, const float *beta,
                                  float *y) {
  return hipsparseScsrmv(handle, transA, static_cast<int>(m),
      static_cast<int>(n), static_cast<int>(nnz), alpha, descrA, val,
      static_cast<const int*>(ptr), static_cast<const int*>(ind), x, beta, y);
}

template <>
hipsparseStatus_t spmv<float, CSC>(hipsparseHandle_t handle,
                                  hipsparseOperation_t transA, INT m, INT n,
                                  INT nnz, const float *alpha,
                                  hipsparseMatDescr_t descrA, const float *val,
                                  const INT *ptr, const INT *ind,
                                  const float *x, const float *beta,
                                  float *y) {
  if (transA ==	HIPSPARSE_OPERATION_TRANSPOSE)
    transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else
    transA = HIPSPARSE_OPERATION_TRANSPOSE;
  return hipsparseScsrmv(handle, transA, static_cast<int>(n),
      static_cast<int>(m), static_cast<int>(nnz), alpha, descrA, val,
      static_cast<const int*>(ptr), static_cast<const int*>(ind), x, beta, y);
}

// 2-Norm
hipblasStatus_t nrm2(hipblasHandle_t handle, INT n, const double *x, INT incx,
                    double *result) {
  return hipblasDnrm2(handle, static_cast<int>(n), x, 
      static_cast<int>(incx), result);
}

hipblasStatus_t nrm2(hipblasHandle_t handle, INT n, const float *x, INT incx,
                    float *result) {
  return hipblasSnrm2(handle, static_cast<int>(n), x,
      static_cast<int>(incx), result);
}

// Axpy
hipblasStatus_t axpy(hipblasHandle_t handle, INT n, double *alpha,
                    const double *x, INT incx, double *y, INT incy) {
  return hipblasDaxpy(handle, static_cast<int>(n), alpha, x,
      static_cast<int>(incx), y, static_cast<int>(incy));
}

hipblasStatus_t axpy(hipblasHandle_t handle, INT n, float *alpha,
                    const float *x, INT incx, float *y, INT incy) {
  return hipblasSaxpy(handle, static_cast<int>(n), alpha, x,
      static_cast<int>(incx), y, static_cast<int>(incy));
}

// Conjugate Gradient Least Squares
template <typename T, CGLS_FMT F>
INT solve(hipsparseHandle_t handle_s, hipblasHandle_t handle_b,
          hipsparseMatDescr_t descr, const T *val, const INT *ptr,
          const INT *ind, const INT m, const INT n, const INT nnz, const T *b,
          T *x, const T shift, const T tol, const INT maxit, bool quiet) {
  // Variable declarations
  T *p, *q, *r, *s;
  T gamma, normp, normq, norms, norms0, normx, xmax;
  char fmt[] = "%5d %9.2e %12.5g\n";
  INT k, flag = 0, indefinite = 0;

  // Constant declarations
  const T kNegOne = static_cast<T>(-1);
  const T kZero = static_cast<T>(0);
  const T kOne = static_cast<T>(1);
  const T kNegShift = static_cast<T>(-shift);
  const T kEps = static_cast<T>(1e-16);

  // Memory Allocation
  hipMalloc(&p, 2 * (m + n) * sizeof(T));
  q = p + n;
  r = q + m;
  s = r + m;
  hipMemcpy(r, b, m * sizeof(T), hipMemcpyDeviceToDevice);
  hipMemcpy(s, x, n * sizeof(T), hipMemcpyDeviceToDevice);

  // r = b - A*x
  spmv<T, F>(handle_s, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, &kNegOne,
      descr, val, ptr, ind, x, &kOne, r);

  // s = A'*r - shift*x
  spmv<T, F>(handle_s, HIPSPARSE_OPERATION_TRANSPOSE, m, n, nnz, &kOne,
      descr, val, ptr, ind, r, &kNegShift, s);

  // Initialize.
  hipMemcpy(p, s, n * sizeof(T), hipMemcpyDeviceToDevice);
  nrm2(handle_b, n, s, 1, &norms);
  hipDeviceSynchronize();
  norms0 = norms;
  gamma = norms0 * norms0;
  nrm2(handle_b, n, x, 1, &normx);
  hipDeviceSynchronize();
  xmax = normx;

  if (!quiet)
    printf("    k     normx        resNE\n");

  for (k = 0; k < maxit && !flag; ++k) {
    // q = A * p
    spmv<T, F>(handle_s, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz, &kOne,
        descr, val, ptr, ind, p, &kZero, q); //checked

    // delta = norm(p)^2 + shift*norm(q)^2
    nrm2(handle_b, n, p, 1, &normp); 
    nrm2(handle_b, m, q, 1, &normq); 
    hipDeviceSynchronize();
    T delta = normq * normq + shift * normp * normp;

    if (delta <= 0)
      indefinite = 1;
    if(delta == 0)
      delta = kEps;
    T alpha = gamma / delta;
    T neg_alpha = -alpha;

    axpy(handle_b, n, &alpha, p, 1, x,  1);
    axpy(handle_b, m, &neg_alpha, q, 1, r,  1);

    // s = A'*r - shift*x
    hipMemcpy(s, x, n * sizeof(T), hipMemcpyDeviceToDevice);
    spmv<T, F>(handle_s, HIPSPARSE_OPERATION_TRANSPOSE, m, n, nnz, &kOne,
        descr, val, ptr, ind, r, &kNegShift, s);

    // norms = norm(s);
    nrm2(handle_b, n, s, 1, &norms);
    hipDeviceSynchronize();

    // gamma = norms^2
    T gamma1 = gamma;
    gamma = norms * norms;
    T beta = gamma / gamma1;

    // p = s + beta*p
    axpy(handle_b, n, &beta, p, 1, s, 1);
    hipMemcpy(p, s, n * sizeof(T), hipMemcpyDeviceToDevice);

    // Convergence
    nrm2(handle_b, n, x, 1, &normx);
    hipDeviceSynchronize();
    xmax = std::max(xmax, normx);
    flag = (norms <= norms0 * tol) || (normx * tol >= 1);
    if (!quiet)
      printf(fmt, k, normx, norms / norms0); 
  }

  // Compute exit status.
  T shrink = normx / xmax;
  if (k == maxit)
    flag = 2;
  else if (indefinite)
    flag = 3;
  else if (shrink * shrink <= tol)
    flag = 4;

  // Free variables and return;
  hipFree(p);
  return flag;
}

// CGLS, with pre-initialized cusparseHandle and cublasHandle.
template <typename T, CGLS_FMT F>
INT solve(hipsparseMatDescr_t descr, const T *val, const INT *ptr, const INT *ind,
          const INT m, const INT n, const INT nnz, const T *b, T *x,
          const T shift, const T tol, const INT maxit, bool quiet) {

  hipsparseHandle_t handle_s;
  hipblasHandle_t handle_b;
  hipsparseCreate(&handle_s);
  hipblasCreate(&handle_b);
  return solve<T, F>(handle_s, handle_b, descr, val, ptr, ind, m, n, nnz, b, x,
      shift, tol, maxit, quiet);
}

// CGLS, with pre-initialized cusparseMatDescr, cusparseHandle and cublasHandle.
template <typename T, CGLS_FMT F>
INT solve(const T *val, const INT *ptr, const INT *ind,
          const INT m, const INT n, const INT nnz, const T *b, T *x,
          const T shift, const T tol, const INT maxit, bool quiet) {

  hipsparseHandle_t handle_s;
  hipblasHandle_t handle_b;
  hipsparseCreate(&handle_s);
  hipblasCreate(&handle_b);
  hipsparseMatDescr_t descr;
  hipsparseCreateMatDescr(&descr);
  return solve<T, F>(handle_s, handle_b, descr, val, ptr, ind, m, n, nnz, b, x,
      shift, tol, maxit, quiet);
}
}  // namespace cgls

