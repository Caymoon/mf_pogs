#include "cml_blas.cuh"

namespace cml {

hipblasStatus_t blas_nrm2(hipblasHandle_t handle, vector<double> *x,
                         double *result) {
  hipblasStatus_t err = hipblasDnrm2(handle, static_cast<int>(x->size), x->data,
      static_cast<int>(x->stride), result);
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_nrm2(hipblasHandle_t handle, vector<float> *x,
                         float *result) {
  hipblasStatus_t err = hipblasSnrm2(handle, static_cast<int>(x->size), x->data,
      static_cast<int>(x->stride), result);
  CublasCheckError(err);
  return err;
}

// Scal.
hipblasStatus_t blas_scal(hipblasHandle_t handle, const double alpha,
                         vector<double> *x) {
  hipblasStatus_t err = hipblasDscal(handle, static_cast<int>(x->size), &alpha,
      x->data, static_cast<int>(x->stride));
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_scal(hipblasHandle_t handle, const float alpha,
                         vector<float> *x) {
  hipblasStatus_t err = hipblasSscal(handle, static_cast<int>(x->size), &alpha,
      x->data, static_cast<int>(x->stride));
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_scal(hipblasHandle_t handle, const double *alpha,
                         vector<double> *x) {
  hipblasStatus_t err = hipblasDscal(handle, static_cast<int>(x->size), alpha,
      x->data, static_cast<int>(x->stride));
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_scal(hipblasHandle_t handle, const float *alpha,
                         vector<float> *x) {
  hipblasStatus_t err = hipblasSscal(handle, static_cast<int>(x->size), alpha,
      x->data, static_cast<int>(x->stride));
  CublasCheckError(err);
  return err;
}

// Asum.
hipblasStatus_t blas_asum(hipblasHandle_t handle, const vector<double> *x,
                         double *result) {
  hipblasStatus_t err = hipblasDasum(handle, x->size, x->data, x->stride, result);
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_asum(hipblasHandle_t handle, const vector<float> *x,
                         float *result) {
  hipblasStatus_t err = hipblasSasum(handle, x->size, x->data, x->stride, result);
  CublasCheckError(err);
  return err;
}

// Dot.
hipblasStatus_t blas_dot(hipblasHandle_t handle, const vector<double> *x,
                        const vector<double> *y, double *result) {
  hipblasStatus_t err = hipblasDdot(handle, static_cast<int>(x->size), x->data,
      static_cast<int>(x->stride), y->data, static_cast<int>(y->stride), result);
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_dot(hipblasHandle_t handle, const vector<float> *x,
                        const vector<float> *y, float *result) {
  hipblasStatus_t err = hipblasSdot(handle, static_cast<int>(x->size), x->data,
      static_cast<int>(x->stride), y->data, static_cast<int>(y->stride), result);
  CublasCheckError(err);
  return err;
}

// Axpy.
hipblasStatus_t blas_axpy(hipblasHandle_t handle, double alpha,
                         const vector<double> *x, vector<double> *y) {
  hipblasStatus_t err = hipblasDaxpy(handle, static_cast<int>(x->size), &alpha,
      x->data, static_cast<int>(x->stride), y->data,
      static_cast<int>(y->stride));
  CublasCheckError(err);
  return err;
}

hipblasStatus_t blas_axpy(hipblasHandle_t handle, float alpha,
                         const vector<float> *x, vector<float> *y) {
  hipblasStatus_t err = hipblasSaxpy(handle, static_cast<int>(x->size), &alpha,
      x->data, static_cast<int>(x->stride), y->data,
      static_cast<int>(y->stride));
  CublasCheckError(err);
  return err;
}

}  // namespace cml

